#include "hip/hip_runtime.h"
#include "cudaConvolution.h"

#include <math.h>
#include <stdio.h>
#include <string.h>

// Size of the blocks on the GPU.  This is the smallest possible
// square block size that is an integer multiple of a warp.  You may
// modify these values if you want.
#define BLOCK_SIZE_X 128
#define BLOCK_SIZE_Y 128

// Size of the stencils.  Do not modify.
#define STENCIL_WIDTH_X 21
#define STENCIL_WIDTH_Y 11

// Global variables to store the convolution stencils.
float *hos_stencil_1dx = NULL;
float *hos_stencil_1dy = NULL;

__constant__ float STENCIL_1DX[STENCIL_WIDTH_X];
__constant__ float STENCIL_1DY[STENCIL_WIDTH_Y];

////////////////////////////////////////////////////////////////
///////////////////////// CUDA kernels /////////////////////////
////////////////////////////////////////////////////////////////
// TO DO: Modify the code in the kernels below to answer the homework
// questions.

__global__ void conv1h_basic_kernel(int width, int height, float *dev_input,
				    float *dev_output) {
  // TODO: This is only an example kernel: it reverses the greyscale
  // value of the input image but does not otherwise modify it.

  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  if ((x < width) && (y < height)) {
    int image_offset = y * width + x;
    int start = image_offset - (STENCIL_WIDTH_X / 2);
    float sum = 0;
    for (int i = 0; i < STENCIL_WIDTH_X; i++) {
      int index = start + i;
      int new_x = x + (index - image_offset);
      if (new_x >= 0 && new_x < width) {
	sum += STENCIL_1DX[i] * dev_input[index];
      }
    }
    dev_output[image_offset] = sum;
  }
}

__global__ void conv1v_basic_kernel(int width, int height, float *dev_input,
				    float *dev_output) {
  // TODO: This is only an example kernel: it reverses the greyscale
  // value of the input image but does not otherwise modify it.

  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  if ((x < width) && (y < height)) {
    int y_offset = (STENCIL_WIDTH_Y / 2);
    float sum = 0;
    int image_offset = y * width + x;
    for (int i = 0; i < STENCIL_WIDTH_Y; i++) {
      int new_y = -y_offset + i + y;
      if (new_y >= 0 && new_y < height) {
	int index = new_y * width + x;
	sum += STENCIL_1DY[i] * dev_input[index];
      }
    }
    dev_output[image_offset] = sum;
  }
}

__global__ void conv1h_tiled_kernel(int width, int height, float *dev_input,
				    float *dev_output) {
  // This code is adapted from the textbook.
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  int i = y * width + x;
  int shared_size = BLOCK_SIZE_X + STENCIL_WIDTH_X - 1;
  __shared__ float N_ds[BLOCK_SIZE_X + STENCIL_WIDTH_X - 1];

  int n = STENCIL_WIDTH_X / 2;

  int dimx = blockDim.x;
  int halo_offsets[4] = {dimx, -dimx}; //, 2*dimx, -2*dimx};

  for (int j = 0; j < 4; j++) {
    int new_x = x + halo_offsets[j];
    int shared_index = threadIdx.x + n + halo_offsets[j];
    if (shared_index < shared_size && shared_index >= 0) {
      int halo_index = y * width + new_x;
      N_ds[shared_index] = (new_x < 0 || new_x >= width) ? 0 : dev_input[halo_index];
    }
  }

  N_ds[n + threadIdx.x] = (x < 0 || x >= width) ? 0 : dev_input[i];

  __syncthreads();

  if ((x < width) && (y < height)) {
    float Pvalue = 0;
    for (int j = 0; j < STENCIL_WIDTH_X; j++) {
      Pvalue += N_ds[threadIdx.x + j] * STENCIL_1DX[j];
    }
    dev_output[i] = Pvalue;
  }
}

__global__ void conv1v_tiled_kernel(int width, int height, float *dev_input,
				    float *dev_output) {
  // This code is adapted from the textbook.
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  int i = y * width + x;
  __shared__ float N_ds[BLOCK_SIZE_Y + STENCIL_WIDTH_Y - 1];

  int n = STENCIL_WIDTH_Y / 2;

  int y_left = y - blockDim.y;
  int halo_index_left = y_left * width + x;
  if (threadIdx.y >= blockDim.y - n) {
    N_ds[threadIdx.y - (blockDim.y - n)] =
      (y_left < 0) ? 0 : dev_input[halo_index_left];
  }

  N_ds[n + threadIdx.y] = (y < 0 || y >= height) ? 0 : dev_input[i];

  int y_right = y + blockDim.y;
  int halo_index_right = y_right * width + x;
  if (threadIdx.y < n) {
    N_ds[n + blockDim.y + threadIdx.y] =
      (y_right >= height) ? 0 : dev_input[halo_index_right];
  }

  __syncthreads();

  if ((x < width) && (y < height)) {
    float Pvalue = 0;
    for (int j = 0; j < STENCIL_WIDTH_Y; j++) {
      Pvalue += N_ds[threadIdx.y + j] * STENCIL_1DY[j];
    }
    dev_output[i] = Pvalue;
  }
}

//////////////////////////////////////////////////////////////////
///////////////////////// Host functions /////////////////////////
//////////////////////////////////////////////////////////////////
// TO DO: Modify the code in the kernels below to answer the homework
// questions.
//
// Notes:
//
// float *hos_stencil_1dx is a host global pointer containing a 1D
// array of length STENCIL_SIZE_X with the stencil data to be used for the
// horizontal convolution.
//
// float *hos_stencil_1dy is a host global pointer containing a 1D
// array of length STENCIL_SIZE_Y with the stencil data to be used for the
// vertical convolution.

void conv1h_basic(int width, int height, float *hos_data_in,
		  float *hos_data_out) {
  // TODO: This host function is mostly complete, but you will need to
  // add some code to set up the constant memory on the device to
  // store the stencil and you may want to modify the grid and block
  // structure for the kernel.

  float *dev_image_in_buffer;
  float *dev_image_out_buffer;

  // Allocate space on the device and copy over the input image.
  int image_size = width * height * sizeof(float);

  hipMemcpyToSymbol(HIP_SYMBOL(STENCIL_1DX), hos_stencil_1dx,
		     STENCIL_WIDTH_X * sizeof(float));

  hipMalloc(&dev_image_in_buffer, image_size);
  hipMalloc(&dev_image_out_buffer, image_size);

  hipMemcpy(dev_image_in_buffer, hos_data_in, image_size,
	     hipMemcpyHostToDevice);

  // Compute grid and block size
  dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
  int grid_size_x = ceil((double)width / BLOCK_SIZE_X);
  int grid_size_y = ceil((double)height / BLOCK_SIZE_Y);
  printf("\ngrid_size (%d, %d)\n", grid_size_x, grid_size_y);
  dim3 gridDim(grid_size_x, grid_size_y, 1);

  // Launch kernel
  conv1h_basic_kernel<<<gridDim, blockDim>>>(width, height, dev_image_in_buffer,
					     dev_image_out_buffer);

  // Synchronization
  hipDeviceSynchronize();

  // Retrieve the output image and free the memory on the device.
  hipMemcpy(hos_data_out, dev_image_out_buffer, image_size,
	     hipMemcpyDeviceToHost);

  hipFree(dev_image_in_buffer);
  hipFree(dev_image_out_buffer);
}

// Q2 (b)
void conv1v_basic(int width, int height, float *hos_data_in,
		  float *hos_data_out) {
  float *dev_image_in_buffer;
  float *dev_image_out_buffer;

  // Allocate space on the device and copy over the input image.
  int image_size = width * height * sizeof(float);

  hipMemcpyToSymbol(HIP_SYMBOL(STENCIL_1DY), hos_stencil_1dy,
		     STENCIL_WIDTH_Y * sizeof(float));

  hipMalloc(&dev_image_in_buffer, image_size);
  hipMalloc(&dev_image_out_buffer, image_size);

  hipMemcpy(dev_image_in_buffer, hos_data_in, image_size,
	     hipMemcpyHostToDevice);

  // Compute grid and block size
  dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
  int grid_size_x = ceil((double)width / BLOCK_SIZE_X);
  int grid_size_y = ceil((double)height / BLOCK_SIZE_Y);
  printf("\ngrid_size (%d, %d)\n", grid_size_x, grid_size_y);
  dim3 gridDim(grid_size_x, grid_size_y, 1);

  // Launch kernel
  conv1v_basic_kernel<<<gridDim, blockDim>>>(width, height, dev_image_in_buffer,
					     dev_image_out_buffer);

  // Synchronization
  hipDeviceSynchronize();

  // Retrieve the output image and free the memory on the device.
  hipMemcpy(hos_data_out, dev_image_out_buffer, image_size,
	     hipMemcpyDeviceToHost);

  hipFree(dev_image_in_buffer);
  hipFree(dev_image_out_buffer);
}

// Q2 (c)
void conv1to2_basic(int width, int height, float *hos_data_in,
		    float *hos_data_out) {
  float *temp = (float *)malloc(width * height * sizeof(float));
  conv1h_basic(width, height, hos_data_in, temp);
  conv1v_basic(width, height, temp, hos_data_out);
  free(temp);
}

// Q3 (a)
void conv1h_tiled(int width, int height, float *hos_data_in,
		  float *hos_data_out) {
  float *dev_image_in_buffer;
  float *dev_image_out_buffer;

  // Allocate space on the device and copy over the input image.
  int image_size = width * height * sizeof(float);

  hipMemcpyToSymbol(HIP_SYMBOL(STENCIL_1DX), hos_stencil_1dx,
		     STENCIL_WIDTH_X * sizeof(float));

  hipMalloc(&dev_image_in_buffer, image_size);
  hipMalloc(&dev_image_out_buffer, image_size);

  hipMemcpy(dev_image_in_buffer, hos_data_in, image_size,
	     hipMemcpyHostToDevice);

  // Compute grid and block size
  dim3 blockDim(BLOCK_SIZE_X, 1, 1);
  int grid_size_x = ceil((double)width / blockDim.x);
  int grid_size_y = ceil((double)height / blockDim.y);
  printf("\ngrid_size (%d, %d)\n", grid_size_x, grid_size_y);
  dim3 gridDim(grid_size_x, grid_size_y, 1);

  // Launch kernel
  conv1h_tiled_kernel<<<gridDim, blockDim>>>(width, height, dev_image_in_buffer,
					     dev_image_out_buffer);

  // Synchronization
  hipDeviceSynchronize();

  // Retrieve the output image and free the memory on the device.
  hipMemcpy(hos_data_out, dev_image_out_buffer, image_size,
	     hipMemcpyDeviceToHost);

  hipFree(dev_image_in_buffer);
  hipFree(dev_image_out_buffer);
}

void conv1v_tiled(int width, int height, float *hos_data_in,
		  float *hos_data_out) {
  float *dev_image_in_buffer;
  float *dev_image_out_buffer;

  // Allocate space on the device and copy over the input image.
  int image_size = width * height * sizeof(float);

  hipMemcpyToSymbol(HIP_SYMBOL(STENCIL_1DY), hos_stencil_1dy,
		     STENCIL_WIDTH_Y * sizeof(float));

  hipMalloc(&dev_image_in_buffer, image_size);
  hipMalloc(&dev_image_out_buffer, image_size);

  hipMemcpy(dev_image_in_buffer, hos_data_in, image_size,
	     hipMemcpyHostToDevice);

  // Compute grid and block size
  dim3 blockDim(1, BLOCK_SIZE_Y, 1);
  int grid_size_x = ceil((double)width / blockDim.x);
  int grid_size_y = ceil((double)height / blockDim.y);
  printf("\ngrid_size (%d, %d)\n", grid_size_x, grid_size_y);
  dim3 gridDim(grid_size_x, grid_size_y, 1);

  // Launch kernel
  conv1v_tiled_kernel<<<gridDim, blockDim>>>(width, height, dev_image_in_buffer,
					     dev_image_out_buffer);

  // Synchronization
  hipDeviceSynchronize();

  // Retrieve the output image and free the memory on the device.
  hipMemcpy(hos_data_out, dev_image_out_buffer, image_size,
	     hipMemcpyDeviceToHost);

  hipFree(dev_image_in_buffer);
  hipFree(dev_image_out_buffer);
}

void conv1to2_tiled(int width, int height, float *hos_data_in,
		    float *hos_data_out) {
  float *temp = (float *)malloc(width * height * sizeof(float));
  conv1h_tiled(width, height, hos_data_in, temp);
  conv1v_tiled(width, height, temp, hos_data_out);
  free(temp);
}

/////////////////////////////////////////////////////////////////////////
///////////////// No change to code after this point ////////////////////
/////////////////////////////////////////////////////////////////////////
// DO NOT CHANGE THE CODE BELOW THIS COMMENT (or in any of the other
// files).  Modification of the code below or in the other files may
// cause the autograder to fail, and you may receive a zero for the
// corresponding questions in the homework.

Convolution::~Convolution() {
  free(hos_stencil_1dx);
  free(hos_stencil_1dy);

  hos_stencil_1dx = hos_stencil_1dy = NULL;
}

void Convolution::evaluate_gaussian_stencil(float sigma, float *stencil_array,
					    int stencil_width) {
  // Compute the stencil
  float normalization = 0.0f;
  int half_width = stencil_width / 2;
  float interval = 20.0f / stencil_width;
  for (int i = 0; i < stencil_width; i++) {
    float x = (i - half_width) * interval;
    float gaussian = std::exp(-(x * x) / (2 * sigma * sigma));
    stencil_array[i] = gaussian;

    normalization += gaussian;
    // printf("%d: %f - %f\n", i, x, gaussian);
  }

  // Normalize so that stencil sums to 1 and store to stencil_array.
  for (int i = 0; i < stencil_width; i++) stencil_array[i] /= normalization;
}

void Convolution::setup_host_gaussian_stencil(float sigma) {
  // Allocate memory, freed in destructor.
  hos_stencil_1dx = (float *)malloc(STENCIL_WIDTH_X * sizeof(float));
  hos_stencil_1dy = (float *)malloc(STENCIL_WIDTH_Y * sizeof(float));

  // Evaluate Gaussian function to create the stencils.
  evaluate_gaussian_stencil(sigma, hos_stencil_1dx, STENCIL_WIDTH_X);
  evaluate_gaussian_stencil(sigma, hos_stencil_1dy, STENCIL_WIDTH_Y);
}

void Convolution::setup_device(int width, int height) {
  hipDeviceProp_t prop;

  int ndev;
  hipGetDeviceCount(&ndev);
  if (ndev < 1) {
    fprintf(stderr, "No CUDA device found\n");
    exit(-1);
  }
  hipGetDeviceProperties(&prop, 0);

  printf("The GPU is a %s\n", prop.name);
  printf("Cuda capability %d.%d.\n", prop.major, prop.minor);
  printf("Shared memory per block %d bytes.\n", prop.sharedMemPerBlock);
}

void Convolution::load_image_input(const Image &image) {
  if (image.channel != 1) {
    printf("Error: Input image has %d channels (should be 1).\n",
	   image.channel);
  }

  int w = image.width, h = image.height;

  // Allocate host input image buffer
  if (!hos_image_in.pixels) {
    hos_image_in.alloc(image.width, image.height, image.channel);
  }

  memcpy(hos_image_in.pixels, image.pixels, w * h * sizeof(float));
  hos_image_out.alloc(w, h, 1);
}

// Since we've copied input to device in function load_image_input,
// we can just launch kernels here:
void Convolution::run_horizontal_1d() {
  int width = hos_image_in.width, height = hos_image_in.height;

  // Call student's code
  conv1h_basic(width, height, hos_image_in.pixels, hos_image_out.pixels);
}

void Convolution::run_vertical_1d() {
  int width = hos_image_in.width, height = hos_image_in.height;

  // Call student's code
  conv1v_basic(width, height, hos_image_in.pixels, hos_image_out.pixels);
}

void Convolution::run_1to2() {
  int width = hos_image_in.width, height = hos_image_in.height;

  conv1to2_basic(width, height, hos_image_in.pixels, hos_image_out.pixels);
}

void Convolution::run_1to2_tiling() {
  int width = hos_image_in.width, height = hos_image_in.height;

  conv1to2_tiled(width, height, hos_image_in.pixels, hos_image_out.pixels);
}

void Convolution::run_2d_tiling() {}

void Convolution::run_horizontal_1d_tiling() {
  int width = hos_image_in.width, height = hos_image_in.height;

  // Call student's code
  conv1h_tiled(width, height, hos_image_in.pixels, hos_image_out.pixels);
}

void Convolution::run_vertical_1d_tiling() {
  int width = hos_image_in.width, height = hos_image_in.height;

  // Call student's code
  conv1v_tiled(width, height, hos_image_in.pixels, hos_image_out.pixels);
}

const float *Convolution::get_image_buffer() { return hos_image_out.pixels; }

const Image *Convolution::get_image_ptr() { return &hos_image_out; }
